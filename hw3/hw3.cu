#include "hip/hip_runtime.h"
/* hw3
 @author Ehsan MohyedinKermani
*/
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include "time_it.h"
#define LOGISTIC 1
#define NORM 2
#define RND 3

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

// struct for passing arguments through time_it_run to our kernel functions in Q1, Q2
struct kernel_arg {
  float *x, *result;
  uint n, m;
  int nblks, tpb;
};

/* Question 1*/

__global__ void logistic(float *x, uint n, uint m) {
	uint i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i < n) {
		float val = x[i];
		for(int iter = 0; iter < m; iter+=64)
			// unrolling for loop 64 times
			val = 3.9f * val * (1.0f - val);
			val = 3.9f * val * (1.0f - val);
			val = 3.9f * val * (1.0f - val);
			val = 3.9f * val * (1.0f - val);
			val = 3.9f * val * (1.0f - val);
			val = 3.9f * val * (1.0f - val);
			val = 3.9f * val * (1.0f - val);
			val = 3.9f * val * (1.0f - val);
			val = 3.9f * val * (1.0f - val);
			val = 3.9f * val * (1.0f - val);
			val = 3.9f * val * (1.0f - val);
			val = 3.9f * val * (1.0f - val);
			val = 3.9f * val * (1.0f - val);
			val = 3.9f * val * (1.0f - val);
			val = 3.9f * val * (1.0f - val);
			val = 3.9f * val * (1.0f - val);
			val = 3.9f * val * (1.0f - val);
			val = 3.9f * val * (1.0f - val);
			val = 3.9f * val * (1.0f - val);
			val = 3.9f * val * (1.0f - val);
			val = 3.9f * val * (1.0f - val);
			val = 3.9f * val * (1.0f - val);
			val = 3.9f * val * (1.0f - val);
			val = 3.9f * val * (1.0f - val);
			val = 3.9f * val * (1.0f - val);
			val = 3.9f * val * (1.0f - val);
			val = 3.9f * val * (1.0f - val);
			val = 3.9f * val * (1.0f - val);
			val = 3.9f * val * (1.0f - val);
			val = 3.9f * val * (1.0f - val);
			val = 3.9f * val * (1.0f - val);
			val = 3.9f * val * (1.0f - val);
			val = 3.9f * val * (1.0f - val);
			val = 3.9f * val * (1.0f - val);
			val = 3.9f * val * (1.0f - val);
			val = 3.9f * val * (1.0f - val);
			val = 3.9f * val * (1.0f - val);
			val = 3.9f * val * (1.0f - val);
			val = 3.9f * val * (1.0f - val);
			val = 3.9f * val * (1.0f - val);
			val = 3.9f * val * (1.0f - val);
			val = 3.9f * val * (1.0f - val);
			val = 3.9f * val * (1.0f - val);
			val = 3.9f * val * (1.0f - val);
			val = 3.9f * val * (1.0f - val);
			val = 3.9f * val * (1.0f - val);
			val = 3.9f * val * (1.0f - val);
			val = 3.9f * val * (1.0f - val);
			val = 3.9f * val * (1.0f - val);
			val = 3.9f * val * (1.0f - val);
			val = 3.9f * val * (1.0f - val);
			val = 3.9f * val * (1.0f - val);
			val = 3.9f * val * (1.0f - val);
			val = 3.9f * val * (1.0f - val);
			val = 3.9f * val * (1.0f - val);
			val = 3.9f * val * (1.0f - val);
			val = 3.9f * val * (1.0f - val);
			val = 3.9f * val * (1.0f - val);
			val = 3.9f * val * (1.0f - val);
			val = 3.9f * val * (1.0f - val);
			val = 3.9f * val * (1.0f - val);
			val = 3.9f * val * (1.0f - val);
			val = 3.9f * val * (1.0f - val);
			val = 3.9f * val * (1.0f - val);
			
	}
}

void do_logistic(void *void_args) {
	struct kernel_arg *argk = (struct kernel_arg *)(void_args);
	logistic<<<argk->nblks, argk->tpb>>>(argk->x, argk->n, argk->m);
	HANDLE_ERROR(hipDeviceSynchronize());
	
	logistic<<<argk->nblks, argk->tpb>>>(argk->x, argk->n, argk->m);
	HANDLE_ERROR(hipDeviceSynchronize());
	
	logistic<<<argk->nblks, argk->tpb>>>(argk->x, argk->n, argk->m);
	HANDLE_ERROR(hipDeviceSynchronize());
	
	logistic<<<argk->nblks, argk->tpb>>>(argk->x, argk->n, argk->m);
	HANDLE_ERROR(hipDeviceSynchronize());
}

void time_logistic(uint n, int m, int tpb, int ntrials) {
	
	int nblks = n / tpb;
	uint size = n * sizeof(float);
	float *x;
	float *dev_x;
	x = (float *)malloc(size);
	// initialize x
	x[0] = 0.123f;
	for(int i = 1; i < n; i++)
		x[i] = 3.9f * x[i - 1] * (1.0f - x[i - 1]);
	
	HANDLE_ERROR(hipMalloc((void**)(&dev_x), size));
	HANDLE_ERROR(hipMemcpy(dev_x, x, size, hipMemcpyHostToDevice));
	
	struct kernel_arg argk;
  	struct time_it_raw *tr = time_it_create(ntrials);
  	struct time_it_stats stats;
	// initialize argk
  	argk.n = n;
  	argk.x = dev_x;
  	argk.m = m;
  	argk.nblks = nblks;
  	argk.tpb = tpb;
  	
  	// run the kernel and report timing info
  	time_it_run(tr, do_logistic, (void *)(&argk));
  	time_it_get_stats(tr, &stats);
  	HANDLE_ERROR(hipMemcpy(x, dev_x, size, hipMemcpyDeviceToHost));
  	printf("Time logistic: mean(T) = %10.3e, std(T) = %10.3e\n", stats.mean, stats.std);
  	printf("Number of GFLOPS is %10.3e\n", (3*m*n/stats.mean)/1e9);
  	free(x);
	HANDLE_ERROR(hipFree(dev_x));
	time_it_free(tr);
}

/* Question 2 */

#define NBLKS 2048
__shared__ float sdata[NBLKS];

__device__ void reduce_sum_dev(float *g_idata, float *g_odata, uint n) {

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    uint tid = threadIdx.x;
    uint i = blockIdx.x * blockDim.x * 2 + threadIdx.x;
    uint gridSize = blockDim.x * 2 * gridDim.x;

    float mySum = 0;

    // we reduce multiple elements per thread.  The number is determined by the
    // number of active thread blocks (via gridDim).  More blocks will result
    // in a larger gridSize and therefore fewer elements per thread
    while (i < n)
    {
        mySum += g_idata[i];
        // ensure we don't read out of bounds
        if (i + blockDim.x < n)
            mySum += g_idata[i+blockDim.x];
        i += gridSize;
    }

    // each thread puts its local sum into shared memory
    sdata[tid] = mySum;
    __syncthreads();


    // do reduction in shared mem
    if (blockDim.x >= 512) { if (tid < 256) { sdata[tid] = mySum = mySum + sdata[tid + 256]; } __syncthreads(); }
    if (blockDim.x >= 256) { if (tid < 128) { sdata[tid] = mySum = mySum + sdata[tid + 128]; } __syncthreads(); }
    if (blockDim.x >= 128) { if (tid <  64) { sdata[tid] = mySum = mySum + sdata[tid +  64]; } __syncthreads(); }

    if (tid < 32)
    {
        // now that we are using warp-synchronous programming (below)
        // we need to declare our shared memory volatile so that the compiler
        // doesn't reorder stores to it and induce incorrect behavior.
        volatile float* smem = sdata;
        if (blockDim.x >=  64) { smem[tid] = mySum = mySum + smem[tid + 32]; }
        if (blockDim.x >=  32) { smem[tid] = mySum = mySum + smem[tid + 16]; }
        if (blockDim.x >=  16) { smem[tid] = mySum = mySum + smem[tid +  8]; }
        if (blockDim.x >=   8) { smem[tid] = mySum = mySum + smem[tid +  4]; }
        if (blockDim.x >=   4) { smem[tid] = mySum = mySum + smem[tid +  2]; }
        if (blockDim.x >=   2) { smem[tid] = mySum = mySum + smem[tid +  1]; }
    }

    // write result for this block to global mem
    if (tid == 0)
        g_odata[blockIdx.x] = sdata[0];
}


__global__ void norm(float *x, float *result, uint n) {
	uint i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i < n)
		x[i] = x[i] * x[i];
	reduce_sum_dev(x, result, n);
}

void do_norm(void *void_args) {
	struct kernel_arg *argk = (struct kernel_arg *)(void_args);
	norm<<<(argk->nblks + argk->tpb - 1)/argk->tpb, argk->tpb>>>(argk->x, argk->result, argk->n);
	HANDLE_ERROR(hipDeviceSynchronize());
}

void mem_norm(uint n, uint nblks, uint tpb, int ntrials) {
	
	uint size = n * sizeof(float);
	float *x, *result;
	float *dev_x, *dev_result;
	x = (float *)malloc(size);
	result = (float *)malloc(size);
	
	x[0] = 0.123f;
	for(int i = 1; i < n; i++)
		x[i] = x[0];
	
	HANDLE_ERROR(hipMalloc((void**)(&dev_x), size));
	HANDLE_ERROR(hipMalloc((void**)(&dev_result), size));
	HANDLE_ERROR(hipMemcpy(dev_x, x, size, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_result, result, size, hipMemcpyHostToDevice));
	struct kernel_arg argk;
  	struct time_it_raw *tr = time_it_create(ntrials);
  	struct time_it_stats stats;
	// initialize argk
  	argk.n = n;
  	argk.x = dev_x;
  	argk.result = dev_result;
  	argk.nblks = nblks;
  	argk.tpb = tpb;
  	
  	// run the kernel and report timing info
  	time_it_run(tr, do_norm, (void *)(&argk));
  	time_it_get_stats(tr, &stats);
  	HANDLE_ERROR(hipMemcpy(x, dev_x, size, hipMemcpyDeviceToHost));
  	HANDLE_ERROR(hipMemcpy(result, dev_result, size, hipMemcpyDeviceToHost));
  	printf("Time norm: mean(T) = %10.3e, std(T) = %10.3e\n", stats.mean, stats.std);
  	printf("Memory-bandwidth is %10.3e, GB/second\n", (n*sizeof(float)/stats.mean)/1e9);
  	free(x);
  	free(result);
	HANDLE_ERROR(hipFree(dev_x));
	HANDLE_ERROR(hipFree(dev_result));
	time_it_free(tr);
}

/* Question 3*/

// struct for passing arguments to time_it_run in Q3
struct kernel_rarg {
	uint nblks, tpb, m, seed;
	uint *x;
	hiprandState *randState;
};

// initialization
__global__ void setup_kernel(uint seed, hiprandState *state) {
  	
  	uint myId = blockDim.x * blockIdx.x + threadIdx.x;
  	seed = blockIdx.x;
  	hiprand_init(seed, myId, 0, &state[myId]);
}
// random number generator
__global__ void rndm(uint *x, uint m, hiprandState *randState) {
	
	uint myId = blockDim.x * blockIdx.x + threadIdx.x;
  	hiprandState *myRandState = &(randState[myId]);
  	for(int j = 0; j < m; j++) {
    	x[myId] = (hiprand_uniform(myRandState) <= 0.5);
  	}
}

void randNumGenerator(void *void_args) {
	struct kernel_rarg *argk = (struct kernel_rarg *)(void_args);
	rndm<<<argk->nblks, argk->tpb>>>(argk->x, argk->m, argk->randState);
	HANDLE_ERROR(hipDeviceSynchronize());
}

void time_randNumGenerator(uint nblks, uint tpb, uint m, int ntrials) {
	
	uint n = nblks * tpb * m; 
	uint size = n * sizeof(uint);
	uint *x;
	uint *dev_x;
	x = (uint *)malloc(size);
	hiprandState *devState;
	
	HANDLE_ERROR(hipMalloc((void**)(&devState), n * sizeof(hiprandState)));
	HANDLE_ERROR(hipMalloc((void**)(&dev_x), size));
	HANDLE_ERROR(hipMemcpy(dev_x, x, size, hipMemcpyHostToDevice));
	struct kernel_rarg argk;
  	struct time_it_raw *tr = time_it_create(ntrials);
  	struct time_it_stats stats;
	// initialize argk
  	argk.x = dev_x;
  	argk.nblks = nblks;
  	argk.tpb = tpb;
  	argk.m = m;
  	argk.randState = devState;
  	
  	// run the kernel and report timing info
  	time_it_run(tr, randNumGenerator, (void *)(&argk));
  	time_it_get_stats(tr, &stats);
  	HANDLE_ERROR(hipMemcpy(x, dev_x, size, hipMemcpyDeviceToHost));
  	printf("Time randNumGenerator: mean(T) = %10.3e, std(T) = %10.3e\n", stats.mean, stats.std);
  	printf("Random number generation speed is %10.3e, random numbers per second\n", (n*sizeof(float)/stats.mean));
  	free(x);
	HANDLE_ERROR(hipFree(dev_x));
	time_it_free(tr);
}

int main(int argc, char **argv) {
	
	uint what = atoi(argv[1]);
	hipDeviceProp_t prop;
	int ndev;
  	HANDLE_ERROR(hipGetDeviceCount(&ndev));
  	if(ndev < 1) {
    	fprintf(stderr, "No CUDA device found!\n");
    	exit(-1);
  	}
  	HANDLE_ERROR(hipGetDeviceProperties(&prop, 0));
  	
  	int sharedMemPerBlock = prop.sharedMemPerBlock;
  	int regsPerBlock = prop.regsPerBlock;
  	printf("GPU is a %s supporing CUDA level %d.%d\n", prop.name, prop.major, prop.minor);
  	printf("It has %d SMs and a warp size of %d\n", prop.multiProcessorCount, prop.warpSize);
  	printf("sharedMemPerBlock = %d, regsPerBlock = %d\n", sharedMemPerBlock, regsPerBlock);
  	printf("clock rate = %d\n", prop.clockRate);
	
	switch(what) {
		case LOGISTIC:
			time_logistic(61440, 6400, 256, 10);
			break;
		case NORM:
			mem_norm(pow(2,26), 6144*2, 256, 10);
			break;
		case RND:
			time_randNumGenerator(1024, 512, 4, 10);
			break;
		default:
      		fprintf(stderr, "ERROR: unknown test case -- %d\n", what);
      		exit(-1);
	}
	exit(0);
}


